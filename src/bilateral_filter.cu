#include "hip/hip_runtime.h"
/*
Mainly 
https://github.com/phrb/intro-cuda/blob/master/src/cuda-samples/3_Imaging/bilateralFilter/bilateral_kernel.cu
*/
#include "bilateral_filter.cuh"
#include <iostream>
#include <stdio.h>

using namespace std;
using namespace cv;

__constant__ float cGaussian[64];
texture<unsigned char, 2, hipReadModeElementType> grayTexture;

extern "C" void setPositionalGaussianDictionary(int radius, float std)
{
    float fGaussian[64];
    for (int i = 0; i < 2 * radius + 1; i++)
    {
        float x = i - radius;
        fGaussian[i] = expf(-(x * x) / (2 * std * std));
    }
    hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float) * (2 * radius + 1));
}

__device__ inline float gaussian(float x, float sigma)
{
    return __expf(-(powf(x, 2)) / (2 * powf(sigma, 2)));
}

__global__ void op_bilateral_filter(unsigned char *input, unsigned char *output,
                                    int width, int height,
                                    int radius, float sigma_pos, float sigma_color)
{
	int x = __mul24(blockIdx.x, 16) + threadIdx.x;
	int y = __mul24(blockIdx.y, 16) + threadIdx.y;

    if (x >= width || y >= height)
    {
        return;
    }

    float total = 0.0f;
    float sum = 0.0f;    
    unsigned char center = tex2D(grayTexture, x, y);
    for (int dy = -radius; dy <= radius; dy++)
    {
        for (int dx = -radius; dx <= radius; dx++)
        {
            unsigned char curPix = tex2D(grayTexture, x + dx, y + dy);
            float weight = cGaussian[dy + radius] * cGaussian[dy + radius] * gaussian(curPix - center, sigma_color);
            total += weight * curPix;
            sum += weight;
        }
    }

    output[y * width + x] = uchar(total/sum);
}

cv::Mat BilateralFilterGPU(const cv::Mat &input_image, const int &radius, const float &sigma_pos, const float& sigma_color)
{
	int gray_size = input_image.step * input_image.rows;
    cv::Mat output_image = Mat::zeros(cv::Size(input_image.cols, input_image.rows), CV_8UC1);

    size_t pitch;
    unsigned char *d_input = NULL;
    unsigned char *d_output;

    setPositionalGaussianDictionary(radius, sigma_pos);

    hipMallocPitch(&d_input, &pitch, sizeof(unsigned char) * input_image.step, input_image.rows);
    hipMemcpy2D(d_input, pitch, input_image.ptr(), sizeof(unsigned char) * input_image.step, sizeof(unsigned char) * input_image.step, input_image.rows, hipMemcpyHostToDevice);
    hipBindTexture2D(0, grayTexture, d_input, input_image.step, input_image.rows, pitch);
    hipMalloc<unsigned char>(&d_output, gray_size);

    dim3 block(16, 16);
    dim3 grid((input_image.cols + block.x - 1) / block.x, (input_image.rows + block.y - 1) / block.y);

    op_bilateral_filter<<<grid, block>>>(d_input, d_output, input_image.cols, input_image.rows, radius, sigma_pos, sigma_color);
    hipMemcpy(output_image.ptr(), d_output, gray_size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output_image;
}